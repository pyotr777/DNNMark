#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <nvToolsExt.h>
#include <nvml.h>
#include <gflags/gflags.h>
#include <glog/logging.h>
#include "warmup.h"


// multiply each element of X to each element of Y and sum
__global__
void multiply(int n, int *x, int *y, int *z) {
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    z[i] = 0;
    for (int j = 0; j < n; j++) {
      z[i] += x[i] * y[j];
    }
  }
}

// Print current GPU state parameters to stdout
void printGPUStateInfo(nvmlDevice_t device, std::string message) {
  nvmlPstates_t pstate;
  nvmlMemory_t memory;
  unsigned int temp;
  unsigned int app_clock;
  unsigned int app_clock_max;
  float clock_perf;
  nvmlReturn_t nvmlRet;

  nvmlRet = nvmlDeviceGetPerformanceState(device, &pstate);
  if (nvmlRet != 0) {
    printf("Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }
  nvmlRet = nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temp);
  if (nvmlRet != 0) {
    printf("Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }
  nvmlRet = nvmlDeviceGetClock(device, NVML_CLOCK_SM, NVML_CLOCK_ID_CURRENT, &app_clock);
  if (nvmlRet != 0) {
    printf("Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }
  nvmlRet = nvmlDeviceGetMaxClockInfo(device, NVML_CLOCK_SM, &app_clock_max);
  if (nvmlRet != 0) {
    printf("Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }
  nvmlRet = nvmlDeviceGetMemoryInfo(device, &memory);
  if (nvmlRet != 0) {
    printf("Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }
  clock_perf = (float)app_clock / (float)app_clock_max * 100.;
  printf("%s P%d, app clock %d/%d MHz (%3.0f%%), %d˚C, memory(free,total): %llu/%llu MB\n",
         message.c_str(), pstate, app_clock, app_clock_max, clock_perf, temp,
         memory.free / 1000000, memory.total / 1000000);
}


/* Call with device number and matrix size */
int warmupGPU(int gpu_id, int iterations, unsigned int size) {
  // bool debug = false;
  nvmlDevice_t nvmldevice;
  nvmlReturn_t nvmlRet;
  std::string message;
  unsigned int N = size;
  int block_size = 256;
  int thread_blocks = (N + block_size - 1) / block_size;

  LOG(INFO) << "blocks: " << thread_blocks << " x " << block_size << ", iterations: "
          << iterations << std::endl;

  int *x, *y, *z;
  // Unified memory allocation
  hipMallocManaged(&x, N * sizeof(int));
  hipMallocManaged(&y, N * sizeof(int));
  hipMallocManaged(&z, N * sizeof(int));

  // initialize x and y arrays on the host
  for (unsigned long i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Init NVML
  nvmlInit();
  LOG(INFO) << "Initialized NVML";
  nvmlRet = nvmlDeviceGetHandleByIndex_v2(gpu_id, &nvmldevice);
  if (nvmlRet != 0) {
    printf("Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }
  message = "Before start:";
  printGPUStateInfo(nvmldevice, message);
  
  auto start = std::chrono::high_resolution_clock::now();

  // Call Warmup procedure
  for (int i = 0; i < iterations; i++) {
    multiply <<< thread_blocks, block_size>>>(N, x, y, z);
  }
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> diff = end - start;
  std::cout << "N=" << N << " " << iterations << " iterations at " << diff.count() * 1e+3 << "ms" << std::endl;

  message = "After warming up:";
  printGPUStateInfo(nvmldevice, message);
  hipFree(x);
  hipFree(y);
  hipFree(z);
  return 0;
}

