#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <nvToolsExt.h>
#include <nvml.h>
#include <gflags/gflags.h>
#include <glog/logging.h>
#include "warmup.h"


// multiply each element of X to each element of Y and sum
__global__
void multiply(int n, int *x, int *y, int *z) {
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    z[i] = 0;
    for (int j = 0; j < n; j++) {
      z[i] += x[i] * y[j];
    }
    // printf("z[%d]=%d, index=%d stride=%d\n", i, z[i], index, stride);
  }
}


// Get GPU clock frequencies
clocks_struct getClocks(nvmlDevice_t device) {
  clocks_struct clocks;
  unsigned int gr_clock = 0;
  unsigned int sm_clock = 0;
  unsigned int sm_clock_max = 0;
  unsigned int mem_clock = 0;
  unsigned int vid_clock = 0;
  float clock_perf = 0;
  nvmlReturn_t nvmlRet;

  nvmlRet = nvmlDeviceGetClock(device, NVML_CLOCK_GRAPHICS, NVML_CLOCK_ID_CURRENT, &gr_clock);
  if (nvmlRet != 0) {
    printf("Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }
  nvmlRet = nvmlDeviceGetClock(device, NVML_CLOCK_SM, NVML_CLOCK_ID_CURRENT, &sm_clock);
  if (nvmlRet != 0) {
    printf("Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }
  nvmlRet = nvmlDeviceGetMaxClockInfo(device, NVML_CLOCK_SM, &sm_clock_max);
  if (nvmlRet != 0) {
    printf("Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }
  nvmlRet = nvmlDeviceGetClock(device, NVML_CLOCK_MEM, NVML_CLOCK_ID_CURRENT, &mem_clock);
  if (nvmlRet != 0) {
    printf("Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }
  nvmlRet = nvmlDeviceGetClock(device, NVML_CLOCK_VIDEO, NVML_CLOCK_ID_CURRENT, &vid_clock);
  if (nvmlRet != 0) {
    printf("Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }  
  clock_perf = (float)sm_clock / (float)sm_clock_max * 100.;

  clocks.gr_clock = gr_clock;
  clocks.sm_clock = sm_clock;
  clocks.sm_clock_max = sm_clock_max;
  clocks.mem_clock = mem_clock;
  clocks.vid_clock = vid_clock;
  clocks.clock_perf = clock_perf;

  return clocks;
}


// Print current GPU state parameters to stdout
void printGPUStateInfo(nvmlDevice_t device, std::string message) {
  nvmlPstates_t pstate;
  nvmlMemory_t memory;
  unsigned int temp;
  clocks_struct clocks;
  nvmlReturn_t nvmlRet;

  clocks = getClocks(device);
  nvmlRet = nvmlDeviceGetPerformanceState(device, &pstate);
  if (nvmlRet != 0) {
    printf("Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }
  nvmlRet = nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temp);
  if (nvmlRet != 0) {
    printf("Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }  
  nvmlRet = nvmlDeviceGetMemoryInfo(device, &memory);
  if (nvmlRet != 0) {
    printf("Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }
  
  LOG(INFO) << message << " P" << pstate << ", smclock " << clocks.clock_perf << "%, " << temp << "˚C"
            << " CLOCKS (graph,sm,mem,vid): " << clocks.gr_clock << "," << clocks.sm_clock << ","
            << clocks.mem_clock << "," << clocks.vid_clock << std::endl;
}


// Main warmup function
void warmup(int FLAGS_warmup, int gpu_id, std::string message) {
  LOG(INFO) << "Warmup function v.1.03";
  if (FLAGS_warmup == 0) {
    return;
  }
  LOG(INFO) << message;
  auto start = std::chrono::high_resolution_clock::now();
  int status = warmupGPU(gpu_id);
  if (status != 0) {
    fprintf(stderr, "Error status: %d\n", status);
  }
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> diff = end - start;
  LOG(INFO) << "Warming up time " + std::to_string(diff.count()*1000) + " ms";
}


/* Call with device number and matrix size */
int warmupGPU(int gpu_id, bool check_results, bool debug) {
  int elements_per_thread = 4;
  float target_warmup = 97.; //% of max app clock Hz
  int maxiter = 100;
  nvmlDevice_t nvmldevice;
  nvmlReturn_t nvmlRet;
  clocks_struct clocks;
  std::string message;
  char deviceName [50];
  hipError_t error;

  // Init NVML
  nvmlRet = nvmlInit_v2();
  if (nvmlRet != 0) {
    printf("NVML init failure. Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }
  nvmlRet = nvmlDeviceGetHandleByIndex_v2(gpu_id, &nvmldevice);
  if (nvmlRet != 0) {
    printf("Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }
  if (debug) {
    message = "Before:";
    printGPUStateInfo(nvmldevice, message);
  }

  // Get GPU properties (Max threads, blocks etc.)
  hipSetDevice(gpu_id);
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop, gpu_id);
  int SMs = dev_prop.multiProcessorCount;
  int SMmax = dev_prop.maxThreadsPerMultiProcessor;
  int max_block_size = dev_prop.maxThreadsPerBlock;
  // get Device name
  nvmlDeviceGetName(nvmldevice, &deviceName[0], 50);
  LOG(INFO) << "GPU " << deviceName << ", " << SMs << " SMs, " << SMmax 
            << " Max threads per SM, " << max_block_size << " max threads per block" << std::endl;


  // Set warmup parameters
  int block_size = max_block_size;
  unsigned int N = SMmax * SMs * elements_per_thread;
  int thread_blocks = (N + block_size - 1) / block_size;

  LOG(INFO) << "Warmup parameters: N=" << N << " elements, " << elements_per_thread 
            << " array elements per thread, "  << thread_blocks << " blocks x "
            << block_size << " threads per block, elements/thread:"
            << elements_per_thread << std::endl;

  int *x, *y, *z, *xd, *yd, *zd;
  x = (int *)malloc(N * sizeof(int));
  y = (int *)malloc(N * sizeof(int));
  z = (int *)malloc(N * sizeof(int));
  hipMalloc(&xd, N * sizeof(int));
  hipMalloc(&yd, N * sizeof(int));
  hipMalloc(&zd, N * sizeof(int));


  // initialize x and y arrays on the host
  for (unsigned long i = 0; i < N; i++) {
    x[i] = 1;
    y[i] = 2;
    z[i] = 0;
  }

  hipMemcpy(xd, x, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(yd, y, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(zd, z, N * sizeof(int), hipMemcpyHostToDevice);

  // Call Warmup procedure
  clocks = getClocks(nvmldevice);
  int i = 1;
  while (clocks.clock_perf < target_warmup and i <= maxiter) {
    auto start = std::chrono::high_resolution_clock::now();
    multiply <<< thread_blocks, block_size>>>(N, xd, yd, zd);
    hipDeviceSynchronize();
    // Wait for GPU to finish before accessing on host
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;
    error = hipGetLastError();
    if (error != 0) {
      std::cout << "CUDA error? " << error << std::endl;
      exit(EXIT_FAILURE);
    }
    // curr_clock = getGPUclock(nvmldevice);
    clocks = getClocks(nvmldevice);
    std::cout << i << "/" << maxiter << " clock " << clocks.clock_perf << "%, time "
              << diff.count() * 1e+3 << "ms"
              << " CLOCKS (graph,sm,mem,vid): " << clocks.gr_clock << "," << clocks.sm_clock << ","
              << clocks.mem_clock << "," << clocks.vid_clock << std::endl;
    i++;
  }
  
  if (check_results) {
    // Check for errors (all values should be 3.0f)
    hipMemcpy(z, zd, N * sizeof(int), hipMemcpyDeviceToHost);
    int maxError = 0;
    unsigned long correct = 2 * N;
    std::cout << "Checking result..." << std::endl;
    for (unsigned long i = 0; i < fmin(N, 10000); i++) {
      maxError = fmax(maxError, fabs(z[i] - correct));
      std::cout << "\r" << i + 1 << "/" << N;
    }
    std::cout << std::endl;
    std::cout << "Max error: " << maxError << std::endl;
  }

  hipFree(xd);
  hipFree(yd);
  hipFree(zd);
  free(x);
  free(y);
  free(z);

  message = "After :";
  printGPUStateInfo(nvmldevice, message);
  // Shutdown NVML
  nvmlRet = nvmlShutdown();
  if (nvmlRet != 0) {
    printf("NVML init failure. Ret: %d\n", nvmlRet);
    exit(EXIT_FAILURE);
  }
  return 0;
}
